#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include ""

using namespace std;

static __device__ void affine_project(float* matrix, float x, float y, float* ox, float* oy) {
	*ox = matrix[0] * x + matrix[1] * y + matrix[2];
	*oy = matrix[3] * x + matrix[4] * y + matrix[5];
}

static __global__ void decode_kernel(
	float* predict, int num_bboxes, int num_classes, float confidence_threshold,
	float* invert_affine_matrix, float* parray, int max_objects, int NUM_BOX_ELEMENT
) {
	int position = blockDim.x * blockIdx.x + threadIdx.x;
	if (position >= num_bboxes) return;

	float* pitem = predict + (5 + num_classes) * position;
	float objectness = pitem[4];
	if (objectness < confidence_threshold)
		return;

	float* class_confidence = pitem + 5;
	float confidence = *class_confidence++;
	int label = 0;
	for (int i = 1; i < num_classes; ++i, ++class_confidence) {
		if (*class_confidence > confidence) {
			confidence = *class_confidence;
			label = i;
		}
	}

	confidence *= objectness;
	if (confidence < confidence_threshold)
		return;

	int index = atomicAdd(parray, 1);
	if (index >= max_objects)
		return;

	float cx = *pitem++;
	float cy = *pitem++;
	float width = *pitem++;
	float height = *pitem++;
	float left = cx - width * 0.5f;
	float top = cy - height * 0.5f;
	float right = cx + width * 0.5f;
	float bottom = cy + height * 0.5f;
	// affine_project(invert_affine_matrix, left,  top,    &left,  &top);
	// affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

	// left, top, right, bottom, confidence, class, keepflag
	float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
	*pout_item++ = left;
	*pout_item++ = top;
	*pout_item++ = right;
	*pout_item++ = bottom;
	*pout_item++ = confidence;
	*pout_item++ = label;
	*pout_item++ = 1; // 1 = keep, 0 = ignore
}

static __device__ float box_iou(
	float aleft, float atop, float aright, float abottom,
	float bleft, float btop, float bright, float bbottom
) {

	float cleft = max(aleft, bleft);
	float ctop = max(atop, btop);
	float cright = min(aright, bright);
	float cbottom = min(abottom, bbottom);

	float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
	if (c_area == 0.0f)
		return 0.0f;

	float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
	float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
	return c_area / (a_area + b_area - c_area);
}

static __global__ void fast_nms_kernel(float* bboxes, int max_objects, float threshold, int NUM_BOX_ELEMENT) {

	int position = (blockDim.x * blockIdx.x + threadIdx.x);
	int count = min((int)*bboxes, max_objects);
	if (position >= count)
		return;

	// left, top, right, bottom, confidence, class, keepflag
	float* pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
	for (int i = 0; i < count; ++i) {
		float* pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
		if (i == position || pcurrent[5] != pitem[5]) continue;

		if (pitem[4] >= pcurrent[4]) {
			if (pitem[4] == pcurrent[4] && i < position)
				continue;

			float iou = box_iou(
				pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
				pitem[0], pitem[1], pitem[2], pitem[3]
			);

			if (iou > threshold) {
				pcurrent[6] = 0;  // 1=keep, 0=ignore
				return;
			}
		}
	}
}

extern "C"
void decode_kernel_invoker(
	float* predict, int num_bboxes, int num_classes, float confidence_threshold,
	float nms_threshold, float* invert_affine_matrix, float* parray, int max_objects, int NUM_BOX_ELEMENT, hipStream_t stream) {

	auto block = num_bboxes > 512 ? 512 : num_bboxes;
	auto grid = (num_bboxes + block - 1) / block;

	/* ����˺����в����ߣ�û��ϵ�����������ģ���ֻ�ǿ���˳�۰��� */
	decode_kernel << <grid, block, 0, stream >> > (
		predict, num_bboxes, num_classes, confidence_threshold,
		invert_affine_matrix, parray, max_objects, NUM_BOX_ELEMENT
		);

	block = max_objects > 512 ? 512 : max_objects;
	grid = (max_objects + block - 1) / block;
	fast_nms_kernel << <grid, block, 0, stream >> > (parray, max_objects, nms_threshold, NUM_BOX_ELEMENT);
}
